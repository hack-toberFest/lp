// #include<bits/stdc++.h>
#include <iostream>
#include <cstdlib> 
#include<hip/hip_runtime.h>
using namespace std;

__global__ void mul(int* A, int* B, int* C, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if(row<N && col<N){
        int ans=0;
        for(int i=0; i<N; i++){
            ans+= A[row*N+i] * B[N*i + col];
        }
        C[row*N+col]=ans;
    }
}

int main(){
    int N=16;
    int size = N * N * sizeof(int);

    int* A;
    int* B;
    int* C;
    int* devA;
    int* devB;
    int* devC;

    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);
    
    hipMalloc(&devA, size);
    hipMalloc(&devB, size);
    hipMalloc(&devC, size);

    for(int i=0; i<N; i++){
        for(int j=0; j<N;j++){
            A[i*N+j]=(rand()%N);
            B[i*N+j]=(rand()%N);
        }
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(16,16);
    dim3 dimGrid(N/dimBlock.x, N/dimBlock.y);

    mul<<<dimGrid, dimBlock>>>(devA, devB, devC, N);

    hipMemcpy(C, devC, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float duration=0;
    hipEventElapsedTime(&duration, start, stop);

    for(int i=0; i<N; i++){
        for(int j=0; j<N;j++){
            cout<<C[i*N+j]<<" ";
        }
        cout<<endl;
    }
    cout<<"time : "<<duration<<" ms"<<endl;

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipEventDestroy(start);
    hipEventDestroy(stop);


}