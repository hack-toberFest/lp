#include<iostream>
#include<cstdlib>
#include<hip/hip_runtime.h>
using namespace std;

__global__ void add(int* A, int*B, int*C, int N){
    int cur = blockIdx.x * blockDim.x + threadIdx.x;

    if(cur<N){
        C[cur]=A[cur]+ B[cur];
    }
}

int main(){
    int N= 10;
    int size = N*sizeof(int);

    int *A, *B, *C;
    int *devA, *devB, *devC;

    hipHostMalloc(&A, size, hipHostMallocDefault);
    hipHostMalloc(&B, size, hipHostMallocDefault);
    hipHostMalloc(&C, size, hipHostMallocDefault);

    hipMalloc(&devA, size);
    hipMalloc(&devB, size);
    hipMalloc(&devC, size);

    srand(time(0));
    for(int i=0; i<N; i++){
        A[i]=(rand()%N);
        B[i]= (rand()%N);
    }

    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    hipMemcpy(devA, A, size, hipMemcpyHostToDevice);
    hipMemcpy(devB, B, size, hipMemcpyHostToDevice);

    int blocksize= 256;
    int numblock = (N+blocksize-1)/blocksize;

    add<<<numblock, blocksize>>>(devA, devB, devC, N);

    hipMemcpy(C, devC, size, hipMemcpyDeviceToHost);

    hipEventRecord(end);
    hipEventSynchronize(end);
    float duration=0;
    hipEventElapsedTime(&duration, start, end);

    for(int i=0; i<N;i++)        cout<<A[i]<<" ";
    cout<<endl;
    for(int i=0; i<N;i++)        cout<<B[i]<<" ";
    cout<<endl;
    for(int i=0; i<N;i++)        cout<<C[i]<<" ";
    cout<<endl;

    cout<<"time : "<<duration<<" ms";

    hipFree(devA);
    hipFree(devB);
    hipFree(devC);
    hipHostFree(A);
    hipHostFree(B);
    hipHostFree(C);
    hipEventDestroy(start);
    hipEventDestroy(end);


}